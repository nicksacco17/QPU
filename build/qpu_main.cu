#include "hip/hip_runtime.h"
/*
 * How to compile (assume cuda is installed at /usr/local/cuda/)
 *   nvcc -c -I/usr/local/cuda/include syevd_example.cpp 
 *   g++ -o a.out syevd_example.o -L/usr/local/cuda/lib64 -lcudart -lcusolver
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hip/hip_complex.h>
#include "../include/Matrix.h"
#include <vector>
#include <complex>
#include <random>
#include <chrono>
#include <iostream>
#include <string>

using std::vector;
using std::complex;
using std::cout;
using std::endl;

int main(int argc, char*argv[])
{
	#if (defined(__CUDA_ARCH__) && (__CUDA_ARCH__ > 0))
		cout << "RUNNING ON GPU" << endl;
	#else
		cout << "RUNNING ON CPU" << endl;
	#endif




	std::chrono::steady_clock::time_point total_start_time = std::chrono::steady_clock::now();

	unsigned int STATE_SIZE = 0;
	unsigned int NUM_QUBITS = 0;
	if (argc >= 2)
	{
		NUM_QUBITS = std::stoi(argv[1]);
		NUM_QUBITS = (NUM_QUBITS < 15) ? NUM_QUBITS : 10;
		STATE_SIZE = std::pow(2, NUM_QUBITS) ;
	}
	else
	{
		NUM_QUBITS = 10;
		STATE_SIZE = 1024;
	}

	cout << "NUMBER QUBITS = " << NUM_QUBITS << ", STATE SIZE = " << STATE_SIZE << endl;
	// STEP 1: BASIC LIBRARY CODE
	std::default_random_engine test_generator;

	//std::uniform_real_distribution<double> distribution(0.0, 1.0);
	std::uniform_int_distribution<int> distribution(0, 100);

	vector<vector<complex<double>>> IN_MAT_RAND(STATE_SIZE, vector<complex<double>>(STATE_SIZE, 0.0));

	for (unsigned int i = 0; i < STATE_SIZE; i++)
	{
		for (unsigned int j = i; j < STATE_SIZE; j++)
		{
			IN_MAT_RAND[i][j] = distribution(test_generator);
			IN_MAT_RAND[j][i] = IN_MAT_RAND[i][j];
		}
	}
	
	Matrix A(IN_MAT_RAND);

	std::chrono::steady_clock::time_point start_time = std::chrono::steady_clock::now();
	A.get_eigenvalues();

	std::chrono::steady_clock::time_point stop_time = std::chrono::steady_clock::now();
	cout << "TOTAL TIME = " << std::chrono::duration_cast<std::chrono::milliseconds>(stop_time - start_time).count() << " msec" << endl;



	return 0;
}
